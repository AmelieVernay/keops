#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include <sstream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "core/pack/Pack.h"
#include "core/pack/GetInds.h"
#include "core/pack/GetDims.h"
#include "core/utils/CudaErrorCheck.cu"
#include "core/utils/CudaSizes.h"
#include "core/mapreduce/GpuConv2D.h"

namespace keops {

struct GpuConv2D_FromHost {
  template < typename TYPE, class FUN >
  static int Eval_(FUN fun, int nx, int ny, TYPE** px_h, TYPE** py_h, TYPE** pp_h) {
    
    using DIMSX = typename FUN::DIMSX;
    using DIMSY = typename FUN::DIMSY;
    using DIMSP = typename FUN::DIMSP;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;
    const int DIMP = DIMSP::SUM;
    const int DIMOUT = FUN::DIM; // dimension of output variable
    const int DIMFOUT = DIMSX::FIRST;     // DIMFOUT is dimension of output variable of inner function
    const int DIMRED = FUN::DIMRED; // dimension of reduction operation
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;
    
    // Compute on device : grid is 2d and block is 1d
    int dev = -1;
    CudaSafeCall(hipGetDevice(&dev));
    
    SetGpuProps(dev);
    
    dim3 blockSize;
    // warning : blockSize.x was previously set to CUDA_BLOCK_SIZE; currently CUDA_BLOCK_SIZE value is used as a bound.
    blockSize.x = ::std::min(CUDA_BLOCK_SIZE,::std::min(maxThreadsPerBlock, (int) (sharedMemPerBlock / ::std::max(1, (int)(DIMY*sizeof(TYPE))) ))); // number of threads in each block
    
    dim3 gridSize;
    gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);
    gridSize.y =  ny / blockSize.x + (ny%blockSize.x==0 ? 0 : 1);
    
    // Reduce  : grid and block are both 1d
    dim3 blockSize2;
    blockSize2.x = CUDA_BLOCK_SIZE; // number of threads in each block
    dim3 gridSize2;
    gridSize2.x =  (nx*DIMRED) / blockSize2.x + ((nx*DIMRED)%blockSize2.x==0 ? 0 : 1);
    
    // Data on the device. We need an "inflated" x1B, which contains gridSize.y "copies" of x_d
    // that will be reduced in the final pass.
    TYPE *x1B, *x_d, *y_d, *param_d;
    
    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;
    
    // single hipMalloc
    void **p_data;
    CudaSafeCall(hipMalloc((void**)&p_data, sizeof(TYPE*)*(SIZEI+SIZEJ+SIZEP)+sizeof(TYPE)*(DIMP+nx*(DIMX-DIMFOUT+DIMOUT)+ny*DIMY+nx*DIMRED*gridSize.y)));
    
    TYPE **p_data_a = (TYPE**)p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;
    p_data_a += SIZEP;
    TYPE *p_data_b = (TYPE*)p_data_a;
    param_d = p_data_b;
    p_data_b += DIMP;
    x_d = p_data_b;
    p_data_b += nx*(DIMX-DIMFOUT+DIMOUT);
    y_d = p_data_b;
    p_data_b += ny*DIMY;
    x1B = p_data_b;
    
    // host arrays of pointers to device data
    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];
    
    // Send data from host to device.
    int nvals;
    
    // if DIMSP is empty (i.e. no parameter), nvals = -1 which could result in a segfault
    if(SIZEP > 0){
      php_d[0] = param_d;
      nvals = DIMSP::VAL(0);
      CudaSafeCall(hipMemcpy(php_d[0], pp_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice));
      
      for(int k=1; k<SIZEP; k++) {
        php_d[k] = php_d[k-1] + nvals;
        nvals = DIMSP::VAL(k);
        CudaSafeCall(hipMemcpy(php_d[k], pp_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice));
      }
    }
    
    phx_d[0] = x_d;
    nvals = nx*DIMOUT;
    for(int k=1; k<SIZEI; k++) {
      phx_d[k] = phx_d[k-1] + nvals;
      nvals = nx*DIMSX::VAL(k);
      CudaSafeCall(hipMemcpy(phx_d[k], px_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice));
    }
    
    // if DIMSY is empty (i.e. no Vj variable), nvals = -1 which could result in a segfault
    if (SIZEJ > 0) {
      phy_d[0] = y_d;
      nvals = ny * DIMSY::VAL(0);
      CudaSafeCall(hipMemcpy(phy_d[0], py_h[0], sizeof(TYPE)*nvals, hipMemcpyHostToDevice));
      for(int k=1; k<SIZEJ; k++) {
        phy_d[k] = phy_d[k-1] + nvals;
        nvals = ny*max(0, (int) DIMSY::VAL(k));
        CudaSafeCall(hipMemcpy(phy_d[k], py_h[k], sizeof(TYPE)*nvals, hipMemcpyHostToDevice));
      }
    }
    
    phx_d[0] = x1B; // we write the result before reduction in the "inflated" vector
    
    // copy arrays of pointers
    CudaSafeCall(hipMemcpy(px_d, phx_d, SIZEI*sizeof(TYPE*), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(py_d, phy_d, SIZEJ*sizeof(TYPE*), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(pp_d, php_d, SIZEP*sizeof(TYPE*), hipMemcpyHostToDevice));
    
    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv2DOnDevice<TYPE><<<gridSize,blockSize,blockSize.x*(DIMY)*sizeof(TYPE)>>>(fun,nx,ny,px_d,py_d,pp_d);
    
    // block until the device has completed
    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();
    
    // Since we've used a 2D scheme, there's still a "blockwise" line reduction to make on
    // the output array px_d[0] = x1B. We go from shape ( gridSize.y * nx, DIMRED ) to (nx, DIMOUT)
    reduce2D<TYPE,DIMRED,DIMOUT,FUN><<<gridSize2, blockSize2>>>(x1B, x_d, px_d, gridSize.y,nx);
    
    // block until the device has completed
    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();
    
    // Send data from device to host.
    CudaSafeCall(hipMemcpy(*px_h, x_d, sizeof(TYPE)*(nx*DIMOUT),hipMemcpyDeviceToHost));
    
    // Free memory.
    CudaSafeCall(hipFree(p_data));
    
    return 0;
  }


// Wrapper around GpuConv2D, which takes lists of arrays *x1, *x2, ..., *y1, *y2, ...
// and use getlist to enroll them into "pointers arrays" px and py.
  template < typename TYPE, class FUN, typename... Args >
  static int Eval(FUN fun, int nx, int ny, int device_id, TYPE* x1_h, Args... args) {
    
    // We set the GPU device on which computations will be performed
    if(device_id!=-1)
      CudaSafeCall(hipSetDevice(device_id));
    
    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;
    
    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;
    
    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;
    
    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;
    
    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];
    
    px_h[0] = x1_h;
    getlist<INDSI>(px_h+1,args...);
    getlist<INDSJ>(py_h,args...);
    getlist<INDSP>(pp_h,args...);
    
    return Eval_(fun,nx,ny,px_h,py_h,pp_h);
    
  }

// same without the device_id argument
  template < typename TYPE, class FUN, typename... Args >
  static int Eval(FUN fun, int nx, int ny, TYPE* x1_h, Args... args) {
    return Eval(fun, nx, ny, -1, x1_h, args...);
  }

// Idem, but with args given as an array of arrays, instead of an explicit list of arrays
  template < typename TYPE, class FUN >
  static int Eval(FUN fun, int nx, int ny, TYPE* x1_h, TYPE** args, int device_id=-1) {
    
    // We set the GPU device on which computations will be performed
    if(device_id!=-1)
      CudaSafeCall(hipSetDevice(device_id));
    
    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;
    
    const int SIZEI = VARSI::SIZE+1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;
    
    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;
    
    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;
    
    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];
    
    px_h[0] = x1_h;
    for(int i=1; i<SIZEI; i++)
      px_h[i] = args[INDSI::VAL(i-1)];
    for(int i=0; i<SIZEJ; i++)
      py_h[i] = args[INDSJ::VAL(i)];
    for(int i=0; i<SIZEP; i++)
      pp_h[i] = args[INDSP::VAL(i)];
    
    return Eval_(fun,nx,ny,px_h,py_h,pp_h);
    
  }
  
  
  
};

}

using namespace keops;

extern "C" int GpuReduc2D_FromHost(int nx, int ny, __TYPE__ *gamma, __TYPE__ **args, int device_id = -1) {
  return Eval< F, GpuConv2D_FromHost >::Run(nx, ny, gamma, args, device_id);
}
