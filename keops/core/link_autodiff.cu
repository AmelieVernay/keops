#include "hip/hip_runtime.h"
#include "core/autodiff.h"
#include "core/GpuConv1D.cu"
#include "core/GpuConv2D.cu"
#include "core/reductions/sum.h"
#include "core/reductions/min.h"
#include "core/reductions/kmin.h"
#include "core/reductions/log_sum_exp.h"

using namespace keops;

///////////////////////////////////////////////
// Convolutions on Gpu device from host data //
///////////////////////////////////////////////

extern "C" int GpuReduc1D(int nx, int ny, __TYPE__* gamma, __TYPE__** args) {
    return Eval<F,GpuConv1D_FromHost>::Run(nx, ny, gamma, args);
}

extern "C" int GpuReduc2D(int nx, int ny, __TYPE__* gamma, __TYPE__** args) {
    return Eval<F,GpuConv2D_FromHost>::Run(nx, ny, gamma, args);
}

//////////////////////////////////////////////////////////
// Convolutions on Gpu device directly from device data //
//////////////////////////////////////////////////////////

extern "C" int GpuReduc1D_FromDevice(int nx, int ny, __TYPE__* gamma, __TYPE__** args) {
    return Eval<F,GpuConv1D_FromDevice>::Run(nx, ny, gamma, args);
}

extern "C" int GpuReduc2D_FromDevice(int nx, int ny, __TYPE__* gamma, __TYPE__** args) {
    return Eval<F,GpuConv2D_FromDevice>::Run(nx, ny, gamma, args);
}

/////////////////////////
// Convolutions on Cpu //
/////////////////////////

#include "core/CpuConv.cpp"

extern "C" int CpuReduc(int nx, int ny, __TYPE__* gamma, __TYPE__** args) {
    return Eval<F,CpuConv>::Run(nx, ny, gamma, args);
}

