#include "hip/hip_runtime.h"

__global__ void GpuConv1DOnDevice(int nx, int ny, {TYPE} *out, {TYPE} **args) {{

  // get the index of the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // declare shared mem
  extern __shared__ {TYPE} yj[];

  // load parameter(s)
  {TYPE} param_loc[{DIMP} < 1 ? 1 : {DIMP}];
  {loadp} //load<DIMSP, INDSP>(0, param_loc, args); // load parameters variables from global memory to local thread memory

  {TYPE} fout[{DIMFOUT}];
  // get the value of variable (index with i)
  {TYPE} xi[{DIMX} < 1 ? 1 : {DIMX}];
  {TYPEACC} acc[{DIMRED}];
  
  if (i < nx) {{
    {InitializeReduction} //<__TYPEACC__, TYPE >()(acc); // acc = 0
    {loadx} //<DIMSX, INDSI>(i, xi, args); // load xi variables from global memory to local thread memory
  }}

  for (int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) {{

    // get the current column
    int j = tile * blockDim.x + threadIdx.x;

    if (j < ny) {{ // we load yj from device global memory only if j<ny
      {loady} //<DIMSY,INDSJ>(j, yj + threadIdx.x * DIMY, args); // load yj variables from global memory to shared memory
    }}
    __syncthreads();

    if (i < nx) {{ // we compute x1i only if needed
      {TYPE} * yjrel = yj; // Loop on the columns of the current block.
      for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++, yjrel += {DIMY}) {{
        {call} //<DIMSX, DIMSY, DIMSP>(fun,fout,xi,yjrel,param_loc); // Call the function, which outputs results in fout
	    {ReducePairShort} //<__TYPEACC__,TYPE>()(acc, fout, jrel + tile * blockDim.x);     // acc += fout
      }}
    }}
    __syncthreads();
  }}
  if (i < nx) {{
    {FinalizeOutput} //<__TYPEACC__,TYPE>()(acc, out + i * DIMOUT, i);
  }}

}}





  extern "C" __host__ int Eval(int nx, int ny, {TYPE} *out {args}) {{

	{TYPE}* args[{nargs}];
	{loadargs}
		  
    // device array of pointers to device data
    {TYPE} **args_d;

    // single hipMalloc
    hipMalloc(&args_d, sizeof({TYPE} *) * {NMINARGS});

    hipMemcpy(args_d, args, {NMINARGS} * sizeof({TYPE} *), hipMemcpyHostToDevice);

    // Compute on device : grid and block are both 1d

    //int dev = -1;
    //hipGetDevice(&dev);

    //SetGpuProps(dev);

    dim3 blockSize;

    blockSize.x = 192;
	
    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

    GpuConv1DOnDevice <<< gridSize, blockSize, blockSize.x * {DIMY} * sizeof({TYPE}) >>> (nx, ny, out, args_d);
    
    // block until the device has completed
    hipDeviceSynchronize();

    //CudaCheckError();

    hipFree(args_d);

    return 0;
  }}
