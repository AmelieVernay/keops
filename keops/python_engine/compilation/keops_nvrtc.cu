
// nvcc -shared -Xcompiler -fPIC -lnvrtc -lcuda keops_nvrtc.cu -o keops_nvrtc.so

#include <hip/hiprtc.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdarg.h>


#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      exit(1);                                                    \
    }                                                             \
  } while(0)
#define CUDA_SAFE_CALL(x)                                         \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      std::cerr << "\nerror: " #x " failed with error "           \
                << msg << '\n';                                   \
      exit(1);                                                    \
    }                                                             \
  } while(0)


char* read_text_file(char const* path) {
    char* buffer = 0;
    long length;
    FILE * f = fopen (path, "rb");
    if (f)
    {
      fseek (f, 0, SEEK_END);
      length = ftell (f);
      fseek (f, 0, SEEK_SET);
      buffer = (char*)malloc ((length+1)*sizeof(char));
      if (buffer)
      {
        fread (buffer, sizeof(char), length, f);
      }
      fclose (f);
    }
    buffer[length] = '\0';
    return buffer;
}


extern "C" __host__ int Compile(const char* ptx_file_name, const char* cu_code) {

    char *ptx;

    hiprtcProgram prog;

    NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog,         // prog
                   cu_code,         // buffer
                   NULL,    // name
                   0,             // numHeaders
                   NULL,          // headers
                   NULL));        // includeNames

    const char *opts[] = {};
    hiprtcResult compileResult = hiprtcCompileProgram(prog,  // prog
                                              0,     // numOptions
                                              opts); // options
              
    // Obtain compilation log from the program.
    size_t logSize;
    NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
    char *log = new char[logSize];
    NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log));
    std::cout << log << '\n';
    delete[] log;
    if (compileResult != HIPRTC_SUCCESS) {
        exit(1);
    }

    // Obtain PTX from the program.
    size_t ptxSize;
    NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
    ptx = new char[ptxSize];
    NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));
    // Destroy the program.
    NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
    
    // write ptx code to file
    FILE *ptx_file = fopen(ptx_file_name, "w");
    fputs(ptx, ptx_file);
    fclose(ptx_file);

    return 0;
}



extern "C" __host__ int launch_keops(const char* ptx_file_name, int dimY, int nx, int ny, int device_id, int tagI, int **ranges, float *out, int nargs, ...) {

    if (tagI==1) {
        int tmp = ny;
        ny = nx;
        nx = tmp;
    }
    
    // reading arguments
    va_list ap;
    va_start(ap, nargs);
    float *arg[nargs];
    for (int i=0; i<nargs; i++)
        arg[i] = va_arg(ap, float*);
    int *argshape[nargs];
    for (int i=0; i<nargs; i++)
        argshape[i] = va_arg(ap, int*);
    va_end(ap);
    
    dim3 blockSize;
    blockSize.x = 32;
	
    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

    
    char *ptx;
    ptx = read_text_file(ptx_file_name);
    hipDevice_t cuDevice;
    hipSetDevice(device_id);
    hipModule_t module;
    hipFunction_t kernel;
    CUDA_SAFE_CALL(hipInit(0));
    CUDA_SAFE_CALL(hipDeviceGet(&cuDevice, device_id));
    CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));
    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, "GpuConv1DOnDevice"));

    void *kernel_params[nargs+3];
    kernel_params[0] = &nx;
    kernel_params[1] = &ny;
    kernel_params[2] = &out;
    for (int i=0; i<nargs; i++)
        kernel_params[i+3] = &arg[i];
    CUDA_SAFE_CALL(hipModuleLaunchKernel(kernel,
                   gridSize.x, gridSize.y, gridSize.z,    // grid dim
                   blockSize.x, blockSize.y, blockSize.z,   // block dim
                   blockSize.x * dimY * sizeof(float), NULL,             // shared mem and stream
                   kernel_params, 0));           // arguments
    CUDA_SAFE_CALL(hipCtxSynchronize());

    CUDA_SAFE_CALL(hipModuleUnload(module));

    return 0;
}
