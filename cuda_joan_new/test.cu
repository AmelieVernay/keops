
// test convolution
// compile with 
//		nvcc -std=c++11 -o test test.cu

 
#define __TYPE__ double
#define __DIMPOINT__ 3 
#define __DIMVECT__ 3 
#define KERNEL SCALARRADIAL 	// type of kernel. Others are VARSURF (varifolds surfaces) and NCSURF (for normal cycles surfaces)
#define EVAL sEval 		// type of convolution for the type of kernel. For scalar radial kernels, others are sGrad1, sGrad, sHess, sDiff 
#define RADIALFUN CauchyFunction 	// Others are GaussFunction, Sum4GaussFunction, Sum4CauchyFunction



#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <vector>

#include "GpuConv2D.cu"

int main()
{
			
    int Nx=10000, Ny=10000;

	
    typedef typename KER::EVAL::DIMSX DIMSX;   
    typedef typename KER::EVAL::DIMSY DIMSY;
    const int SIZEX = DIMSX::SIZE;
    const int SIZEY = DIMSY::SIZE;        

    __TYPE__ *x[SIZEX];

    __TYPE__ *y[SIZEY];

	vector<__TYPE__>* tmp;
	
	// create random inputs
	for(int k=0; k<SIZEX; k++)
	{
		tmp = new vector<__TYPE__>(Nx*DIMSX::VAL(k));	
		x[k] = tmp->data();	
		for(int i=0; i<Nx*DIMSX::VAL(k); i++)
			x[k][i] = ((__TYPE__)rand())/RAND_MAX;
	}			
	
	for(int k=0; k<SIZEY; k++)
	{
		tmp = new vector<__TYPE__>(Ny*DIMSY::VAL(k));
		y[k] = tmp->data();
		for(int i=0; i<Ny*DIMSY::VAL(k); i++)
			y[k][i] = ((__TYPE__)rand())/RAND_MAX;
	}

	// set GPU device number
    //int deviceID = 0;
	//hipSetDevice(deviceID);

	struct KER::EVAL funeval;
	typedef RADIALFUN<__TYPE__> RadialFun;

	// compute
	//GpuConv2D(KER(RadialFun()),funeval,Nx,Ny,x,y);
	GpuConv2D(KER(RadialFun()),funeval,Nx,Ny,x[0],x[1],y[0],y[1]);
	
	// display sum of output values
	__TYPE__ s = 0;
	for(int i=0; i<Nx*DIMSX::VAL(0); i++)
		s += x[0][i];
	cout << "sum=" << s << endl;

}

