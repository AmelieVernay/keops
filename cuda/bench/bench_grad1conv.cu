#include "hip/hip_runtime.h"
#include <iostream>
#include <benchmark/benchmark.h>

// use manual timing for GPU based functions
#include <chrono>
#include <ctime>

#include "bench/generate_data.h"

#include "core/GpuConv1D.cu"
#include "core/GpuConv2D.cu"
#include "core/CpuConv.cpp"

#include "core/formulas/constants.h"
#include "core/formulas/maths.h"
#include "core/formulas/kernels.h"
#include "core/formulas/norms.h"
#include "core/formulas/factorize.h"

#include "core/autodiff.h"

using namespace std;

/////////////////////////////////////////////////////////////////////////////////////
//                      The function to be benchmarked                            //
/////////////////////////////////////////////////////////////////////////////////////

#define F0 Grad<GaussKernel<_P<0>,_X<0,3>,_Y<1,3>,_Y<2,3>>,_X<0,3>,_X<3,3>>
using FUN0 = typename Generic<F0>::sEval;
#define F1 F0
using FUN1 = typename Generic<F1>::sEval;

extern "C" int GaussGpuGrad1Conv(__TYPE__ ooSigma2, __TYPE__* alpha_h, __TYPE__* x_h, __TYPE__* y_h, __TYPE__* beta_h, __TYPE__* gamma_h, int dimPoint, int dimVect, int nx, int ny) ;



/////////////////////////////////////////////////////////////////////////////////////
//                                The Bench                                        //
/////////////////////////////////////////////////////////////////////////////////////


// The zeroth benchmark : simply to avoid to warm up the GPU...
static void BM_dummy(benchmark::State& state) {
    for (auto _ : state) {
        int Nx =100;
        
        data<__TYPE__> data1(Nx);

        vector<__TYPE__> vf0(Nx*data1.dimPoint);  __TYPE__ *rescpu  = vf0.data(); 
        vector<__TYPE__> vf2(Nx*data1.dimPoint);  __TYPE__ *resgrad2 = vf2.data(); 
        vector<__TYPE__> vf3(Nx*data1.dimPoint);  __TYPE__ *resgrad1 = vf3.data(); 
        vector<__TYPE__> vf4(Nx*data1.dimPoint);  __TYPE__ *resgpu2 = vf4.data(); 
        vector<__TYPE__> vf5(Nx*data1.dimPoint);  __TYPE__ *resgpu1 = vf5.data(); 

        GaussGpuGrad1Conv(data1.params[0], data1.u, data1.x, data1.y, data1.v, rescpu, data1.dimPoint,data1.dimVect,data1.Nx,data1.Ny); 
        GpuConv2D(FUN0(), data1.params, data1.Nx, data1.Ny, resgrad2, data1.x, data1.y, data1.v, data1.u);
        GpuConv1D(FUN0(), data1.params, data1.Nx, data1.Ny, resgrad1, data1.x, data1.y, data1.v, data1.u);
        GpuConv2D(FUN1(), data1.params, data1.Nx, data1.Ny, resgpu2, data1.x, data1.y, data1.v, data1.u);
        GpuConv1D(FUN1(), data1.params, data1.Nx, data1.Ny, resgpu1, data1.x, data1.y, data1.v, data1.u);

        // display values
        cout << endl << "specific  = ";
        for(int i=0; i<10; i++)
            cout << rescpu[i] << " ";
        cout << endl << "gpugrad2d = ";
        for(int i=0; i<10; i++)
            cout << resgrad2[i] << " ";
        cout << endl << "gpugrad1d = ";
        for(int i=0; i<10; i++)
            cout << resgrad1[i] << " ";
        cout << endl << "gpugen2d = ";
        for(int i=0; i<10; i++)
            cout << resgpu2[i] << " ";
        cout << endl << "gpugen1d = ";
        for(int i=0; i<10; i++)
            cout << resgpu1[i] << " ";

        __TYPE__ e=0;
        for (int i=0; i<Nx*data1.dimPoint; i++){
            e+= abs(rescpu[i] - resgrad1[i]) ;
        }
        cout << endl << "Erreur (specific vs gpugrad1d) : " << e << endl;

        e=0;
        for (int i=0; i<Nx*data1.dimPoint; i++){
            e+= abs(resgrad2[i] - rescpu[i]) ;
        }
        cout << "Erreur (specific vs gpugrad2d) : " << e << endl;
        
        e=0;
        for (int i=0; i<Nx*data1.dimPoint; i++){
            e+= abs(resgrad2[i] - resgrad1[i]) ;
        }
        cout << "Erreur (gpugrad1d vs gpugrad2d) : " << e << endl << endl;
    }
}
BENCHMARK(BM_dummy);// Register the function as a benchmark


/////////////////////////////////////////////////////////////////////////////////////////////////
void main_grad_1D(int Nx) {
    data<__TYPE__> data1(Nx);
    GpuConv1D(FUN0(), data1.params, data1.Nx, data1.Ny, data1.f, data1.x, data1.y, data1.v, data1.u);
}

static void cuda_grad_1D(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_grad_1D(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start);
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_grad_1D)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark




/////////////////////////////////////////////////////////////////////////////////////////////////
void main_grad_2D(int Nx) {
    data<__TYPE__> data1(Nx);
    GpuConv2D(FUN0(), data1.params, data1.Nx, data1.Ny, data1.f, data1.x, data1.y, data1.v, data1.u);
}

static void cuda_grad_2D(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_grad_2D(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start);
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_grad_2D)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark




/////////////////////////////////////////////////////////////////////////////////////////////////
void main_generic_1D(int Nx) {
    data<__TYPE__> data1(Nx);
    GpuConv1D(FUN1(), data1.params, data1.Nx, data1.Ny, data1.f, data1.x, data1.y, data1.v, data1.u);
}

static void cuda_generic_1D(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_generic_1D(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start);
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_generic_1D)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark




/////////////////////////////////////////////////////////////////////////////////////////////////
void main_generic_2D(int Nx) {
    data<__TYPE__> data1(Nx);
    GpuConv2D(FUN1(), data1.params, data1.Nx, data1.Ny, data1.f, data1.x, data1.y, data1.v, data1.u);
}

static void cuda_generic_2D(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_generic_2D(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start);
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_generic_2D)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark




/////////////////////////////////////////////////////////////////////////////////////////////////

void main_specific(int Nx) {
    data<__TYPE__> data1(Nx);
    GaussGpuGrad1Conv(data1.params[0], data1.u, data1.x, data1.y, data1.v, data1.f, data1.dimPoint,data1.dimVect,data1.Nx,data1.Ny); 
}

static void cuda_specific(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_specific(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start);
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_specific)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark




BENCHMARK_MAIN();// generate the benchmarks
