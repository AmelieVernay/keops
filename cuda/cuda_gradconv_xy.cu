#include "hip/hip_runtime.h"
/*
*	This cuda routine allows one to compute the derivative wrt the point cloud 'y' of the derivative
*	wrt 'x' of the expression
*		K(x_i,y_j) @ b_j =  sum_j f( |x_i-y_j|^2 ) b_j
*	
*	
*	We're looking for the gradient with respect to y of
*	
*	< e, K(s,a,x,y,b) >  =  \sum_{i,j} f_s'( |x_i-y_j|^2 ) * < a_i, b_j > * 2 < e_i, x_i-y_j>,
*	
*	which is an M-by-D array g_j (j from 1 to M), where each line is equal to
*	
*	g_j = -2* \sum_i < a_i, b_j > * [                       f_s'(  |x_i-y_j|^2 ) * e_i
*	                                + 2* < x_i-y_j, e_i > * f_s''( |x_i-y_j|^2 ) * (x_i-y_j) ]
* 
*	We will compute this sum over the index 'i' on the GPU, with 'one thread' = 'one index j'.
*	Data will be stored as follow:
*	  - e_i in the SharedData
* 	  - a_i in the SharedData
*	  - x_i in the SharedData
*	  - y_j in the thread memory
*	  - b_j in the thread memory
* 
* 
* Author : Jean Feydy, heavily based on the work of Joan Glaunès and Benjamin Charlier.
* 
*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "kernels_old.cx"


#define UseCudaOnDoubles USE_DOUBLE_PRECISION

///////////////////////////////////////
/////////// CUDA KERNEL ///////////////
///////////////////////////////////////


template < typename TYPE, int DIMPOINT, int DIMVECT > // Typically, float32, D, E
__global__ void GaussGpuGradConvXYOnDevice(TYPE ooSigma2, // 1/sigma^2
		TYPE *e,                                   // N-by-D array
		TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, // N-by-E, N-by-D, M-by-D, M-by-E arrays
		TYPE *gamma,                               // Output variable, M-by-D (same as y)
		int nx, int ny)
{
    // Thread kernel:
    // Computation of gamma_j = \partial_{y_j} < e_i, \partial_{x_i} < alpha_i, sum_j k(x_i,y_j)*beta_j > >
    //
    //                        = -2* \sum_i < a_i, b_j > * [                       f_s'(  |x_i-y_j|^2 ) * e_i
    //                                                    + 2* < x_i-y_j, e_i > * f_s''( |x_i-y_j|^2 ) * (x_i-y_j) ]
    // for index j given by thread id.
    
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // the following line does not work with nvcc 3.0 (it is a bug; it works with anterior and posterior versions)
    // extern __shared__ TYPE SharedData[];  // shared data will contain x and alpha data for the block
    // here is the bug fix (see http://forums.nvidia.com/index.php?showtopic=166905)
    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);
    // end of bug fix
    
    // One thread = One line = One y_j + One b_j + One gamma_j + a whole bunch of "e_i", "a_i", "x_i".
    TYPE yj[DIMPOINT], betaj[DIMVECT], xmy[DIMPOINT], gammaj[DIMPOINT];
    if(j<ny) { // we will compute gammaj only if j is in the range
        for(int k=0; k<DIMPOINT; k++)
            yj[k]     =     y[j*DIMPOINT+k]; // load y_j from device global memory
        for(int k=0; k<DIMVECT; k++)
            betaj[k]  =  beta[j*DIMVECT+k];  // load b_j from device global memory
        for(int k=0; k<DIMPOINT; k++)    // output : M-by-D : DIMPOINT
            gammaj[k] = 0.0f;            // Make sure to put to zero the output array 
    }

    // Here, we use a tiled matrix decomposition. See cuda_conv.cu for graphs and explanations.
    // Note that here, each thread reads from top to bottom (i++), instead of left to right (j++):
    for(int istart = 0, tile = 0; istart < nx; istart += blockDim.x, tile++) {

        // Load data in Shared memory -----------------------------------------------------------
        int i = tile * blockDim.x + threadIdx.x; // Current line
        // We load ei, alphai and xi from device global memory...
        if(i<nx) { // ...only if i<nx (we may be in the last columns of the last tile...)
            // Pretty uneasy to read : we store ei, ai and xi interleaved, for better performance
            // SharedData = "[ e0, a0, x0, e1, a1, x1, e2, a2, x2 ... ]"
            int inc = DIMPOINT + DIMVECT + DIMPOINT; // Size of a  [ei, ai, xi] block
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k]                  =     e[i*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k]         = alpha[i*DIMVECT +k];
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+DIMVECT+k] =     x[i*DIMPOINT+k];
        }
        __syncthreads();
        // At this point :
        // - y_j, b_j sit in the thread memory
        // - [e_I, ..., e_{I+blockDim.x}], [a_I, ..., a_{I+blockDim.x}] and [x_I, ..., x_{I+blockDim.x}] sit
        //   in the SharedData, where [I : I+blockDim.x] is the tile span.
        // - the output line gamma_j is in the thread memory, and contains the result
        //   of the summation over the previous tiles.
      
        
        // Map-Reduction loop -------------------------------------------------------------------
        // We can now proceed to the "tiled" matrix product, where one line = one thread.
        if(j<ny) // we compute gammaj only if j is in the range
        {
            TYPE *ei, *alphai, *xi;           // As ei, ai and xi are interleaved...
            ei     = SharedData;              // We'll on some cute pointer arithmetics!
            alphai = SharedData + DIMPOINT;
            xi     = SharedData + DIMPOINT + DIMVECT;
            int inc = DIMPOINT  + DIMVECT + DIMPOINT; // The increment, size of a [ei, ai, xi] block.
            
            for(int irel = 0; irel < blockDim.x && irel<nx-istart; irel++, ei+=inc, alphai+=inc, xi+=inc) {
                // Reduction loop over i : we're getting to the maths ***************************
                // Remember: we're computing 
                //    g_j  = -2* \sum_i < a_i, b_j > * [                       f_s'(  |x_i-y_j|^2 ) * e_i
                //                                     + 2* < x_i-y_j, e_i > * f_s''( |x_i-y_j|^2 ) * (x_i-y_j) ]

                TYPE r2 = 0.0f, ei_s_xmy = 0.0f, ai_s_bj;
                // Compute x_i-y_j and its squared norm:
                for(int k=0; k<DIMPOINT; k++) {
                    xmy[k]  =  xi[k]-yj[k];
                    r2     += xmy[k]*xmy[k];
                }
                // Compute < e_i, x_i-y_j> :
                for(int k=0; k<DIMPOINT; k++) // Scalar product between POINTS.
                    ei_s_xmy += ei[k]*xmy[k];
                // Compute < a_i, b_j> :
                for(int k=0; k<DIMVECT; k++)  // Scalar product between VECTORS.
                    ai_s_bj  += alphai[k]* betaj[k];
                // Scalar factor for the first line,   "-2* <a_i,b_j> * f_s'( |x_i-y_j|^2 )"
                TYPE s1 =  -2.0f * ai_s_bj *            GaussFp(  r2 , ooSigma2 );
                // Scalar factor for the second line,  "-4* <a_i,b_j> * < e_i, x_i-y_j > * f_s''( |x_i-y_j|^2 )"
                TYPE s2 =  -4.0f * ai_s_bj * ei_s_xmy * GaussFpp( r2 , ooSigma2 );
                
                for(int k=0; k<DIMPOINT; k++)    // Output: M-by-D
                    gammaj[k] += s1 * ei[k] + s2 * xmy[k];  // Final increment
                // ******************************************************************************
            }
        }
        // Once the loop is over, the current tiled matrix product has been reduced to gamma_j
        __syncthreads();  // So make sure that no one's left behind...
        // And move on to the next tile.
    }

    // Save the result in global memory.
    if(j<ny)
        for(int k=0; k<DIMPOINT; k++)        // Remember: the output, here, is M-by-D (-> DIMPOINT)
            gamma[j*DIMPOINT+k] = gammaj[k];
}

//////////////////////////////////////////////////////
/////////// CPU -> GPU -> CPU routines ///////////////
//////////////////////////////////////////////////////


#if !(UseCudaOnDoubles) 
extern "C" int GaussGpuGradConvXY(float ooSigma2,               // 1 / sigma^2
								float* e_h,                     // N-by-D array (same as x)
								float* alpha_h, float* x_h,     // N-by-E, N-by-D arrays
								float* y_h,     float* beta_h,  // M-by-D, M-by-E arrays
								float* gamma_h,                 // Output: M-by-D (same as y)
								int dimPoint, int dimVect, int nx, int ny){ // D, E, N, M

	// Data on the device.
	float* e_d;
	float* alpha_d;
	float* x_d;
	float* y_d;
	float* beta_d;
	float* gamma_d;

	// Allocate arrays on device.
	hipMalloc((void**)&e_d,     sizeof(float)*(nx*dimPoint));
	hipMalloc((void**)&alpha_d, sizeof(float)*(nx*dimVect ));
	hipMalloc((void**)&x_d,     sizeof(float)*(nx*dimPoint));
	hipMalloc((void**)&y_d,     sizeof(float)*(ny*dimPoint));
	hipMalloc((void**)&beta_d,  sizeof(float)*(ny*dimVect ));
	hipMalloc((void**)&gamma_d, sizeof(float)*(ny*dimPoint)); // Output: M-by-D (same as y)

	// Send data from host to device.
	hipMemcpy(e_d,     e_h,     sizeof(float)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(alpha_d, alpha_h, sizeof(float)*(nx*dimVect ), hipMemcpyHostToDevice);
	hipMemcpy(x_d,     x_h,     sizeof(float)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(y_d,     y_h,     sizeof(float)*(ny*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(beta_d,  beta_h,  sizeof(float)*(ny*dimVect ), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  ny / blockSize.x + (ny%blockSize.x==0 ? 0 : 1); // NB: here, we're working columnwise !

	// Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time : 
	// NB: Here, we use more SharedData than in the rowwise code !
	//     One block of SharedData = [ei,ai,xi], of size (dimPoint+dimVect+dimPoint)*sizeof(float)
	if(     dimPoint==1 && dimVect==1)
		GaussGpuGradConvXYOnDevice<float,1,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==1)
		GaussGpuGradConvXYOnDevice<float,2,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==1)
		GaussGpuGradConvXYOnDevice<float,3,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==1)
		GaussGpuGradConvXYOnDevice<float,4,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==2)
		GaussGpuGradConvXYOnDevice<float,2,2><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==3)
		GaussGpuGradConvXYOnDevice<float,3,3><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==4)
		GaussGpuGradConvXYOnDevice<float,4,4><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else
	{
		printf("GaussGpuGradConvXY error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_gradconv_xy.cu file !");
		hipFree(e_d);
		hipFree(alpha_d);
		hipFree(x_d);
		hipFree(y_d);
		hipFree(beta_d);
		hipFree(gamma_d);
		return(-1);
	}

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(float)*(ny*dimPoint),hipMemcpyDeviceToHost); // Output: M-by-D (same as y)

	// Free memory.
	hipFree(e_d);
	hipFree(alpha_d);
	hipFree(x_d);
	hipFree(y_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}

#else
//////////////////////////////////////////////////////////////
extern "C" int GaussGpuGradConvXY(double ooSigma2,               // 1 / sigma^2
								double* e_h,                     // N-by-D array (same as x)
								double* alpha_h, double* x_h,     // N-by-E, N-by-D arrays
								double* y_h,     double* beta_h,  // M-by-D, M-by-E arrays
								double* gamma_h,                 // Output: M-by-D (same as y)
								int dimPoint, int dimVect, int nx, int ny){ // D, E, N, M

	// Data on the device.
	double* e_d;
	double* alpha_d;
	double* x_d;
	double* y_d;
	double* beta_d;
	double* gamma_d;

	// Allocate arrays on device.
	hipMalloc((void**)&e_d,     sizeof(double)*(nx*dimPoint));
	hipMalloc((void**)&alpha_d, sizeof(double)*(nx*dimVect ));
	hipMalloc((void**)&x_d,     sizeof(double)*(nx*dimPoint));
	hipMalloc((void**)&y_d,     sizeof(double)*(ny*dimPoint));
	hipMalloc((void**)&beta_d,  sizeof(double)*(ny*dimVect ));
	hipMalloc((void**)&gamma_d, sizeof(double)*(ny*dimPoint)); // Output: M-by-D (same as y)

	// Send data from host to device.
	hipMemcpy(e_d,     e_h,     sizeof(double)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(alpha_d, alpha_h, sizeof(double)*(nx*dimVect ), hipMemcpyHostToDevice);
	hipMemcpy(x_d,     x_h,     sizeof(double)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(y_d,     y_h,     sizeof(double)*(ny*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(beta_d,  beta_h,  sizeof(double)*(ny*dimVect ), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  ny / blockSize.x + (ny%blockSize.x==0 ? 0 : 1); // NB: here, we're working columnwise !

	// Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time : 
	// NB: Here, we use more SharedData than in the rowwise code !
	//     One block of SharedData = [ei,ai,xi], of size (dimPoint+dimVect+dimPoint)*sizeof(double)
	if(     dimPoint==1 && dimVect==1)
		GaussGpuGradConvXYOnDevice<double,1,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==1)
		GaussGpuGradConvXYOnDevice<double,2,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==1)
		GaussGpuGradConvXYOnDevice<double,3,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==1)
		GaussGpuGradConvXYOnDevice<double,4,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==2)
		GaussGpuGradConvXYOnDevice<double,2,2><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==3)
		GaussGpuGradConvXYOnDevice<double,3,3><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==4)
		GaussGpuGradConvXYOnDevice<double,4,4><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect+dimPoint)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else
	{
		printf("GaussGpuGradConvXY error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_gradconv_xy.cu file !");
		hipFree(e_d);
		hipFree(alpha_d);
		hipFree(x_d);
		hipFree(y_d);
		hipFree(beta_d);
		hipFree(gamma_d);
		return(-1);
	}

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(double)*(ny*dimPoint),hipMemcpyDeviceToHost); // Output: M-by-D (same as y)

	// Free memory.
	hipFree(e_d);
	hipFree(alpha_d);
	hipFree(x_d);
	hipFree(y_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}

#endif

void ExitFcn(void)
{
    hipDeviceReset();
}
