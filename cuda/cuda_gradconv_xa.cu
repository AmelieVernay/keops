#include "hip/hip_runtime.h"
/*
*	This cuda routine allows one to compute the derivative wrt the dual vector 'a' of the derivative
*	wrt 'x' of the expression
*		K(x_i,y_j) @ b_j =  sum_j f( |x_i-y_j|^2 ) b_j
*	
*	
*	We're looking for the gradient with respect to a of
*	
*	< e, K(s,a,x,y,b) >  =  \sum_{i,j} f_s'( |x_i-y_j|^2 ) * < a_i, b_j > * 2 < e_i, x_i-y_j>,
*	
*	which is an N-by-E array g_i (i from 1 to N), where each line is equal to
*	
*	g_i  =  \sum_j 2* f_s'( |x_i-y_j|^2 ) * < e_i, x_i-y_j> * b_j
*	
*	We will compute this sum over the index 'j' on the GPU, with 'one thread' = 'one index i'.
*	Data will be stored as follow:
*	  - e_i in the thread memory
*	  - x_i in the thread memory
*	  - y_j in the SharedData
*	  - b_j in the SharedData (beta_j, really)
* 
* 
* Author : Jean Feydy, heavily based on the work of Joan Glaunès and Benjamin Charlier.
* 
*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "radial_kernels.cx"


#define UseCudaOnDoubles USE_DOUBLE_PRECISION

///////////////////////////////////////
/////////// CUDA KERNEL ///////////////
///////////////////////////////////////
#if !(UseCudaOnDoubles) 
typedef  float(*KernelFun)( float,  float);
#else
typedef double(*KernelFun)(double, double);
#endif

template < typename TYPE, int DIMPOINT, int DIMVECT, KernelFun KernelFp > // Typically, float32, D, E, GaussFp
__global__ void KernelGpuGradConvXAOnDevice(TYPE ooSigma2, // 1/sigma^2
		TYPE *e,                                   // N-by-D array
		TYPE *alpha, TYPE *x, TYPE *y, TYPE *beta, // N-by-E, N-by-D, M-by-D, M-by-E arrays
		TYPE *gamma,                               // Output variable, N-by-E (same as alpha)
		int nx, int ny)
{
    // Thread kernel:
    // Computation of gamma_i = \partial_{alpha_i} < e_i, \partial_{x_i} < alpha_i, sum_j k(x_i,y_j)*beta_j > >
    //                        = \sum_j 2* f_s'( |x_i-y_j|^2 ) * < e_i, x_i-y_j> * b_j
    // for index i given by thread id.
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // the following line does not work with nvcc 3.0 (it is a bug; it works with anterior and posterior versions)
    // extern __shared__ TYPE SharedData[];  // shared data will contain x and alpha data for the block
    // here is the bug fix (see http://forums.nvidia.com/index.php?showtopic=166905)
    extern __shared__ char SharedData_char[];
    TYPE* const SharedData = reinterpret_cast<TYPE*>(SharedData_char);
    // end of bug fix
    
    // One thread = One line = One x_i + One e_i + One gamma_i + a whole bunch of "y_j", "b_j".
    TYPE ei[DIMPOINT], xi[DIMPOINT], xmy[DIMPOINT], gammai[DIMVECT];
    if(i<nx) { // we will compute gammai only if i is in the range
        for(int k=0; k<DIMPOINT; k++)
            ei[k]     = e[i*DIMPOINT+k]; // load e_i from device global memory
        for(int k=0; k<DIMPOINT; k++)
            xi[k]     = x[i*DIMPOINT+k]; // load x_i from device global memory
        for(int k=0; k<DIMVECT; k++)     // output : N-by-E : DIMVECT
            gammai[k] = 0.0f;            // Make sure to put to zero the output array 
    }

    // Here, we use a tiled matrix decomposition. See cuda_conv.cu for graphs and explanations.
    
    for(int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) {

        // Load data in Shared memory -----------------------------------------------------------
        int j = tile * blockDim.x + threadIdx.x; // Current column
        // We load yj and betaj from device global memory...
        if(j<ny) { // ...only if j<ny (we may be in the last columns of the last tile...)
            // Pretty uneasy to read : we store yj and betaj interleaved, for better performance
            // SharedData = "[ y0, b0, y1, b1, y2, b2, ... ]"
            int inc = DIMPOINT + DIMVECT; // Size of a  [yj, bj] block
            for(int k=0; k<DIMPOINT; k++)
                SharedData[threadIdx.x*inc+k]          =    y[j*DIMPOINT+k];
            for(int k=0; k<DIMVECT; k++)
                SharedData[threadIdx.x*inc+DIMPOINT+k] = beta[j*DIMVECT+k];
        }
        __syncthreads();
        // At this point :
        // - e_i, x_i sit in the thread memory
        // - [y_N, ..., y_{N+blockDim.x}] and [b_N, ..., b_{N+blockDim.x}] sit
        //   in the SharedData, where [N : N+blockDim.x] is the tile span.
        // - the output line gamma_i is in the thread memory, and contains the result
        //   of the summation over the previous tiles.
      
        
        // Map-Reduction loop -------------------------------------------------------------------
        // We can now proceed to the "tiled" matrix product, where one line = one thread.
        if(i<nx) // we compute gammai only if i is in the range
        {
            TYPE *yj, *betaj;                  // As y_j and beta_j are interleaved...
            yj      = SharedData;              // We'll on some cute pointer arithmetics!
            betaj   = SharedData + DIMPOINT;
            int inc = DIMPOINT   + DIMVECT;    // The increment, size of a [y_j,b_j] block.
            
            for(int jrel = 0; jrel < blockDim.x && jrel<ny-jstart; jrel++, yj+=inc, betaj+=inc) {
                // Reduction loop over j : we're getting to the maths ***************************
                // Remember: we're computing 
                //        g_i  =  \sum_j 2* f_s'( |x_i-y_j|^2 ) * < e_i, x_i-y_j> * b_j

                TYPE r2 = 0.0f, ei_s_xmy = 0.0f;
                // Compute x_i-y_j and its squared norm:
                for(int k=0; k<DIMPOINT; k++) {
                    xmy[k]  =  xi[k]-yj[k];
                    r2     += xmy[k]*xmy[k];
                }
                // Compute < e_i, x_i-y_j> :
                for(int k=0; k<DIMPOINT; k++) // Scalar product between POINTS.
                    ei_s_xmy += ei[k]*xmy[k];
                // Scalar factor,   "2* f_s'( |x_i-y_j|^2 ) * < e_i, x_i-y_j>"
                TYPE s =  2.0f * ei_s_xmy * KernelFp( r2 , ooSigma2 );
                for(int k=0; k<DIMVECT; k++)    // Output: N-by-E
                    gammai[k] += s * betaj[k];  // Final increment
                // ******************************************************************************
            }
        }
        // Once the loop is over, the current tiled matrix product has been reduced to gamma_i
        __syncthreads();  // So make sure that no one's left behind...
        // And move on to the next tile.
    }

    // Save the result in global memory.
    if(i<nx)
        for(int k=0; k<DIMVECT; k++)        // Remember: the output, here, is N-by-E (-> DIMVECT)
            gamma[i*DIMVECT+k] = gammai[k];
}

//////////////////////////////////////////////////////
/////////// CPU -> GPU -> CPU routines ///////////////
//////////////////////////////////////////////////////


#if !(UseCudaOnDoubles) 
template < KernelFun KernelFp >
int KernelGpuGradConvXA(float ooSigma2,               // 1 / sigma^2
								float* e_h,                     // N-by-D array (same as x)
								float* alpha_h, float* x_h,     // N-by-E, N-by-D arrays
								float* y_h,     float* beta_h,  // M-by-D, M-by-E arrays
								float* gamma_h,                 // Output: N-by-E (same as alpha)
								int dimPoint, int dimVect, int nx, int ny){ // D, E, N, M

	// Data on the device.
	float* e_d;
	float* alpha_d;
	float* x_d;
	float* y_d;
	float* beta_d;
	float* gamma_d;

	// Allocate arrays on device.
	hipMalloc((void**)&e_d,     sizeof(float)*(nx*dimPoint));
	hipMalloc((void**)&alpha_d, sizeof(float)*(nx*dimVect ));
	hipMalloc((void**)&x_d,     sizeof(float)*(nx*dimPoint));
	hipMalloc((void**)&y_d,     sizeof(float)*(ny*dimPoint));
	hipMalloc((void**)&beta_d,  sizeof(float)*(ny*dimVect ));
	hipMalloc((void**)&gamma_d, sizeof(float)*(nx*dimVect )); // Output: N-by-E (same as alpha)

	// Send data from host to device.
	hipMemcpy(e_d,     e_h,     sizeof(float)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(alpha_d, alpha_h, sizeof(float)*(nx*dimVect ), hipMemcpyHostToDevice);
	hipMemcpy(x_d,     x_h,     sizeof(float)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(y_d,     y_h,     sizeof(float)*(ny*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(beta_d,  beta_h,  sizeof(float)*(ny*dimVect ), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	// Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time : 
	if(     dimPoint==1 && dimVect==1)
		KernelGpuGradConvXAOnDevice<float,1,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==1)
		KernelGpuGradConvXAOnDevice<float,2,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==1)
		KernelGpuGradConvXAOnDevice<float,3,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==1)
		KernelGpuGradConvXAOnDevice<float,4,1,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==2)
		KernelGpuGradConvXAOnDevice<float,2,2,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==3)
		KernelGpuGradConvXAOnDevice<float,3,3,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==4)
		KernelGpuGradConvXAOnDevice<float,4,4,KernelFp><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(float)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else
	{
		printf("GaussGpuGradConvXA error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_gradconv_xa.cu file !");
		hipFree(e_d);
		hipFree(alpha_d);
		hipFree(x_d);
		hipFree(y_d);
		hipFree(beta_d);
		hipFree(gamma_d);
		return(-1);
	}

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(float)*(nx*dimVect),hipMemcpyDeviceToHost); // Output: N-by-E (same as alpha)

	// Free memory.
	hipFree(e_d);
	hipFree(alpha_d);
	hipFree(x_d);
	hipFree(y_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}


// Couldn't find a clean way to give a name to an explicit instantiation :-(
extern "C" int GaussGpuGradConvXA(float ooSigma2, float* e_h,
                                float* alpha_h, float* x_h, float* y_h, float* beta_h, float* gamma_h,
                                int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<GaussFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int LaplaceGpuGradConvXA(float ooSigma2, float* e_h,
                                float* alpha_h, float* x_h, float* y_h, float* beta_h, float* gamma_h,
                                int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<LaplaceFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}
extern "C" int EnergyGpuGradConvXA(float ooSigma2, float* e_h,
                                float* alpha_h, float* x_h, float* y_h, float* beta_h, float* gamma_h,
                                int dimPoint, int dimVect, int nx, int ny) {
    return KernelGpuGradConvXA<EnergyFp>(ooSigma2, e_h, alpha_h, x_h, y_h, beta_h, gamma_h, dimPoint, dimVect, nx, ny);
}

#else
//////////////////////////////////////////////////////////////
extern "C" int GaussGpuGradConvXA(double ooSigma2,               // 1 / sigma^2
								double* e_h,                     // N-by-D array (same as x)
								double* alpha_h, double* x_h,    // N-by-E, N-by-D arrays
								double* y_h,     double* beta_h, // M-by-D, M-by-E arrays
								double* gamma_h,                 // Output: N-by-E (same as alpha)
								int dimPoint, int dimVect, int nx, int ny){ // D, E, N, M

	// Data on the device.
	double* e_d;
	double* alpha_d;
	double* x_d;
	double* y_d;
	double* beta_d;
	double* gamma_d;

	// Allocate arrays on device.
	hipMalloc((void**)&e_d,     sizeof(double)*(nx*dimPoint));
	hipMalloc((void**)&alpha_d, sizeof(double)*(nx*dimVect ));
	hipMalloc((void**)&x_d,     sizeof(double)*(nx*dimPoint));
	hipMalloc((void**)&y_d,     sizeof(double)*(ny*dimPoint));
	hipMalloc((void**)&beta_d,  sizeof(double)*(ny*dimVect ));
	hipMalloc((void**)&gamma_d, sizeof(double)*(nx*dimVect )); // Output: N-by-E (same as alpha)

	// Send data from host to device.
	hipMemcpy(e_d,     e_h,     sizeof(double)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(alpha_d, alpha_h, sizeof(double)*(nx*dimVect ), hipMemcpyHostToDevice);
	hipMemcpy(x_d,     x_h,     sizeof(double)*(nx*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(y_d,     y_h,     sizeof(double)*(ny*dimPoint), hipMemcpyHostToDevice);
	hipMemcpy(beta_d,  beta_h,  sizeof(double)*(ny*dimVect ), hipMemcpyHostToDevice);

	// compute on device.
	dim3 blockSize;
	blockSize.x = CUDA_BLOCK_SIZE; // number of threads in each block
	dim3 gridSize;
	gridSize.x =  nx / blockSize.x + (nx%blockSize.x==0 ? 0 : 1);

	// Copy-paste templating, allowing us to pass the DIMPOINT and DIMVECT at compilation time : 
	if(     dimPoint==1 && dimVect==1)
		GaussGpuGradConvXAOnDevice<double,1,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==1)
		GaussGpuGradConvXAOnDevice<double,2,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==1)
		GaussGpuGradConvXAOnDevice<double,3,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==1)
		GaussGpuGradConvXAOnDevice<double,4,1><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==2 && dimVect==2)
		GaussGpuGradConvXAOnDevice<double,2,2><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==3 && dimVect==3)
		GaussGpuGradConvXAOnDevice<double,3,3><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else if(dimPoint==4 && dimVect==4)
		GaussGpuGradConvXAOnDevice<double,4,4><<<gridSize,blockSize,blockSize.x*(dimPoint+dimVect)*sizeof(double)>>>
			(ooSigma2, e_d, alpha_d, x_d, y_d, beta_d, gamma_d, nx, ny);
	else
	{
		printf("GaussGpuGradConvXA error: dimensions of Gauss kernel not implemented in cuda\nYou probably just need a copy-paste in the conda_gradconv_xa.cu file !");
		hipFree(e_d);
		hipFree(alpha_d);
		hipFree(x_d);
		hipFree(y_d);
		hipFree(beta_d);
		hipFree(gamma_d);
		return(-1);
	}

	// block until the device has completed
	hipDeviceSynchronize();

	// Send data from device to host.
	hipMemcpy(gamma_h, gamma_d, sizeof(double)*(nx*dimVect),hipMemcpyDeviceToHost); // Output: N-by-E (same as alpha)

	// Free memory.
	hipFree(e_d);
	hipFree(alpha_d);
	hipFree(x_d);
	hipFree(y_d);
	hipFree(beta_d);
	hipFree(gamma_d);

	return 0;
}
#endif

void ExitFcn(void)
{
    hipDeviceReset();
}
