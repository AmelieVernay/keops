#include "hip/hip_runtime.h"
// test convolution with autodiff
// compile with
//		nvcc -std=c++11 -O2 -o build/test_autodiff test_autodiff.cu

// we define an arbitrary function using available blocks,
// then test its convolution on the GPU, then get its gradient and test again the convolution

// Here we build the function F(x,y,u,v,beta) = <u,v>^2 * exp(-C*|x-y|^2) * beta
// where x, y, beta are 3D vectors, and u, v are 4D vectors
// and the convolution is gamma_i = sum_j F(x_i,y_j,u_i,v_j,beta_j)
// then we define G(x,y,u,v,beta,eta) = gradient of F with respect to x, with new input variable eta (3D)
// and the new convolution is gamma_i = sum_j G(x_i,y_j,u_i,v_j,beta_j,eta_i)

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ctime>
#include <algorithm>

#include "../core/GpuConv1D.cu"
#include "../core/GpuConv2D.cu"

#define __TYPE__ float

#include "../core/autodiff.h"

#include "../core/CpuConv.cpp"

using namespace std;



__TYPE__ floatrand() {
    return ((__TYPE__)rand())/RAND_MAX-.5;    // random value between -.5 and .5
}

template < class V > void fillrandom(V& v) {
    generate(v.begin(), v.end(), floatrand);    // fills vector with random values
}

int main() {
    // In this part we define the symbolic variables of the function
    using X = Var<0,3,0>; 	// X is the first variable and represents a 3D vector
    using Y = Var<1,3,1>; 	// Y is the second variable and represents a 3D vector
    using U = Var<2,4,0>; 	// U is the third variable and represents a 4D vector
    using V = Var<3,4,1>; 	// V is the fourth variable and represents a 4D vector
    using Beta = Var<4,3,1>;	// Beta is the fifth variable and represents a 3D vector
    using C = Param<0>;		// C is the first extra parameter

    // symbolic expression of the function ------------------------------------------------------
    
    // here we define F = <U,V>^2 * exp(-C*|X-Y|^2) * Beta in usual notations
    using F = Scal<Square<Scalprod<U,V>>, Scal<Exp<Scal<Constant<C>,Minus<SqNorm2<Subtract<X,Y>>>>>,Beta>>;

    using FUNCONVF = typename Generic<F>::sEval;

    // gradient with respect to X ---------------------------------------------------------------
    using Eta = Var<5,F::DIM,0>; // new variable is in sixth position and is input of gradient
    using GX = Grad<F,X,Eta>;
    
    /*
     * Using GX = Grad<F,X,Eta> = (\partial_X F).Eta in a convolution sum (Generic<...>) makes sense.
     * Indeed, we know that
     * 
     *      FUNCONVF_i = \sum_j F( P, X^0_i, X^1_i, ..., Y^0_j, Y^1_j, ... ).
     * 
     * Then, since FUNCONVF_i only depends on the i-th line of X^n,
     * 
     * (\partial_{X^n} FUNCONVF).Eta = \sum_i (\partial_{X^n  } FUNCONVF_i).Eta_i       (definition of the L2 scalar product)
     * 
     *                                        | 0 0 ................................. 0 |
     *                                        | 0 0 ................................. 0 |
     *                               = \sum_i |  (\partial_{X^n_i} FUNCONVF_i).Eta_i    | <- (on the i-th line).
     *                                        | 0 0 ................................. 0 |
     *                                        | 0 0 ................................. 0 |
     *                                        | 0 0 ................................. 0 |
     * 
     *                                        |  (\partial_{X^n_0} FUNCONVF_0).Eta_0    |
     *                                        |  (\partial_{X^n_1} FUNCONVF_1).Eta_1    |
     *                               =        |                    .                    | 
     *                                        |                    .                    |
     *                                        |                    .                    |
     *                                        |  (\partial_{X^n_I} FUNCONVF_I).Eta_I    |
     * 
     * But then, by linearity of the gradient operator,
     * 
     * (\partial_{X^n_i} FUNCONVF_i).Eta_i = \sum_j (\partial_{X^n} F( P, X^0_i, ..., Y^0_j, ... )).Eta_i
     * 
     * (\partial_{X^n} FUNCONVF).Eta is therefore equal to the "generic kernel product" with
     * summation on j, with the summation term being
     * 
     *    (\partial_{X^n_i} F( P, X^0_i, ..., Y^0_j, ... )).Eta_i  = Grad<F,X^n,Eta>
     * 
     */
    using FUNCONVGX = typename Generic<GX>::sEval;

    // gradient with respect to Y  --------------------------------------------------------------
    using GY = Grad<F,Y,Eta>;
    
    /*
     * Using GY = Grad<F,Y,Eta> = (\partial_Y F).Eta in a convolution sum (Generic<...>) makes sense...
     * IF YOU CHANGE THE SUMMATION VARIABLE FROM j TO i !
     * Indeed, we know that
     * 
     *      FUNCONVF_i = \sum_j F( P, X^0_i, X^1_i, ..., Y^0_j, Y^1_j, ... ).
     * 
     * Hence, doing the computations :
     * 
     * (\partial_{Y^m} FUNCONVF).Eta 
     *    = \sum_i    (\partial_{Y^m  } FUNCONVF_i).Eta_i                          (definition of the L2 scalar product)
     *    = \sum_i    (\partial_{Y^m  } \sum_j F(P,X^0_i, ...,Y^0_j,...) ).Eta_i   (FUNCONVF_i = ...)
     *    = \sum_j    \sum_i (\partial_{Y^m  } F(P,X^0_i, ...,Y^0_j,...) ).Eta_i   (Fubini theorem + linearity of \partial_{Y^M})
     * 
     *              | 0 0 .................................................... 0 | (the summation term only depends on Y^m_j)
     *              | 0 0 .................................................... 0 |
     *    = \sum_j  | \sum_i (\partial_{Y^m_j} F(P,X^0_i, ...,Y^0_j,...) ).Eta_i | <- (on the j-th line)
     *              | 0 0 .................................................... 0 |
     *              | 0 0 .................................................... 0 |
     *              | 0 0 .................................................... 0 |
     *              | 0 0 .................................................... 0 |
     * 
     *              | \sum_i (\partial_{Y^m_0} F(P,X^0_i, ...,Y^0_0,...) ).Eta_i |
     *              | \sum_i (\partial_{Y^m_1} F(P,X^0_i, ...,Y^0_1,...) ).Eta_i |
     *    =         |                               .                            | 
     *              |                               .                            | 
     *              |                               .                            | 
     *              |                               .                            | 
     *              | \sum_i (\partial_{Y^m_J} F(P,X^0_i, ...,Y^0_J,...) ).Eta_i |
     * 
     * 
     * (\partial_{Y^m} FUNCONVF).Eta is therefore equal to the "generic kernel product" with
     * summation on i (and not j !), with the summation term being
     * 
     *    (\partial_{Y^m_j} F( P, X^0_i, ..., Y^0_j, ... )).Eta_i  = Grad<F,Y^m,Eta>
     * 
     */
    // parameter 1 after GY means i and j variables must be swapped, 
    // i.e. we do a summation on "i" using a code which is hardcoded for summation wrt. "j" :
    using FUNCONVGY = typename Generic<GY,1>::sEval;

    // now we test ------------------------------------------------------------------------------

    int Nx=5000, Ny=2000;

    vector<__TYPE__> vf(Nx*F::DIM);    fillrandom(vf); __TYPE__ *f = vf.data();
    vector<__TYPE__> vx(Nx*X::DIM);    fillrandom(vx); __TYPE__ *x = vx.data();
    vector<__TYPE__> vy(Ny*Y::DIM);    fillrandom(vy); __TYPE__ *y = vy.data();
    vector<__TYPE__> vu(Nx*U::DIM);    fillrandom(vu); __TYPE__ *u = vu.data();
    vector<__TYPE__> vv(Ny*V::DIM);    fillrandom(vv); __TYPE__ *v = vv.data();
    vector<__TYPE__> vb(Ny*Beta::DIM); fillrandom(vb); __TYPE__ *b = vb.data();

    vector<float> resgpu2D(Nx*F::DIM), resgpu1D(Nx*F::DIM), rescpu(Nx*F::DIM);

    __TYPE__ params[1];
    __TYPE__ Sigma = 1;
    params[0] = 1.0/(Sigma*Sigma);

    clock_t begin, end;

    begin = clock();
    int deviceID = 0;
    hipSetDevice(deviceID);
    end = clock();
    cout << "time for GPU initialization : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    cout << "testing function F" << endl;
    begin = clock();
    GpuConv2D(FUNCONVF(), params, Nx, Ny, f, x, y, u, v, b);
    end = clock();
    cout << "time for GPU computation (first run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    resgpu2D = vf;

    begin = clock();
    GpuConv1D(FUNCONVF(), params, Nx, Ny, f, x, y, u, v, b);
    end = clock();
    cout << "time for GPU computation (second run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    resgpu1D = vf;

    begin = clock();
    CpuConv(FUNCONVF(), params, Nx, Ny, f, x, y, u, v, b);
    end = clock();
    cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    rescpu = vf;

    // display mean of errors
    __TYPE__ s = 0;
    for(int i=0; i<Nx*F::DIM; i++)
        s += abs(resgpu2D[i]-rescpu[i]);
    cout << "mean abs error 2D =" << s/Nx << endl;

    s = 0;
    for(int i=0; i<Nx*F::DIM; i++)
        s += abs(resgpu1D[i]-rescpu[i]);
    cout << "mean abs error 1D =" << s/Nx << endl;




    vector<__TYPE__> ve(Nx*Eta::DIM); fillrandom(ve); __TYPE__ *e = ve.data();

    cout << "testing function GX" << endl;
    begin = clock();
    GpuConv2D(FUNCONVGX(), params, Nx, Ny, f, x, y, u, v, b, e);
    end = clock();
    cout << "time for GPU computation (first run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    resgpu2D = vf;

    begin = clock();
    GpuConv1D(FUNCONVGX(), params, Nx, Ny, f, x, y, u, v, b, e);
    end = clock();
    cout << "time for GPU computation (second run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    resgpu1D = vf;

    begin = clock();
    CpuConv(FUNCONVGX(), params, Nx, Ny, f, x, y, u, v, b, e);
    end = clock();
    cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    rescpu = vf;

    // display mean of errors
    s = 0;
    for(int i=0; i<Nx*GX::DIM; i++)
        s += abs(resgpu2D[i]-rescpu[i]);
    cout << "mean abs error 2D =" << s/Nx << endl;

    s = 0;
    for(int i=0; i<Nx*GX::DIM; i++)
        s += abs(resgpu1D[i]-rescpu[i]);
    cout << "mean abs error 1D =" << s/Nx << endl;



    // gradient wrt Y, which is a "j" variable.

    rescpu.resize(Ny*GY::DIM); resgpu2D.resize(Ny*GY::DIM); resgpu1D.resize(Ny*GY::DIM);
    vf.resize(Ny*GY::DIM);
    f = vf.data();

    cout << "testing function GY" << endl;
    begin = clock();
    GpuConv2D(FUNCONVGY(), params, Ny, Nx, f, x, y, u, v, b, e);
    end = clock();
    cout << "time for GPU computation (first run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    resgpu2D = vf;

    begin = clock();
    GpuConv1D(FUNCONVGY(), params, Ny, Nx, f, x, y, u, v, b, e);
    end = clock();
    cout << "time for GPU computation (second run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    resgpu1D = vf;

    begin = clock();
    CpuConv(FUNCONVGY(), params, Ny, Nx, f, x, y, u, v, b, e);
    end = clock();
    cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << endl;

    rescpu = vf;

    // display mean of errors
    s = 0;
    for(int i=0; i<Ny*GY::DIM; i++)
        s += abs(resgpu2D[i]-rescpu[i]);
    cout << "mean abs error 2D=" << s/Ny << endl;

    s = 0;
    for(int i=0; i<Ny*GY::DIM; i++)
        s += abs(resgpu1D[i]-rescpu[i]);
    cout << "mean abs error 1D=" << s/Ny << endl;



}



