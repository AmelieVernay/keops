#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <benchmark/benchmark.h>

// use manuaml timing for GPU based functions
#include <chrono>
#include <ctime>

using namespace std;

/////////////////////////////////////////////////////////////////////////////////////
//                      The function to be benchmarked                            //
/////////////////////////////////////////////////////////////////////////////////////

// Some convenient functions
float floatrand() {
    return ((float)rand())/RAND_MAX-.5;    // random value between -.5 and .5
}

template < class V > void fillrandom(V& v) {
    generate(v.begin(), v.end(), floatrand);    // fills vector with random values
}

// Signature of the generic function:
extern "C" int GpuConv2D(float*, int, int, float*, float**);

void main_generic_2D(int Nx) {

    int Ny= Nx /2 ;

    int dimPoint = 3;
    int dimVect = 3;

    vector<float> vf(Nx*dimPoint);
    fillrandom(vf);
    float *f = vf.data();

    vector<float> vx(Nx*dimPoint);
    fillrandom(vx);
    float *x = vx.data();
    
    vector<float> vy(Ny*dimPoint);
    fillrandom(vy);
    float *y = vy.data();
    
    vector<float> vu(Nx*dimVect);
    fillrandom(vu);
    float *u = vu.data();
    
    vector<float> vv(Ny*dimVect);
    fillrandom(vv);
    float *v = vv.data();

    // wrap variables
    vector<float*> vargs(4);
    vargs[0]=x;
    vargs[1]=y;
    vargs[2]=v;
    vargs[3]=u;
    float **args = vargs.data();

    float params[1];
    float Sigma = 1;
    params[0] = 1.0/(Sigma*Sigma);

    GpuConv2D(params, Nx, Ny, f, args);

}

// Signature of the generic function:
extern "C" int GpuConv1D(float*, int, int, float*, float**);

void main_generic_1D(int Nx) {

    int Ny= Nx /2 ;

    int dimPoint = 3;
    int dimVect = 3;

    vector<float> vf(Nx*dimPoint);
    fillrandom(vf);
    float *f = vf.data();

    vector<float> vx(Nx*dimPoint);
    fillrandom(vx);
    float *x = vx.data();
    
    vector<float> vy(Ny*dimPoint);
    fillrandom(vy);
    float *y = vy.data();
    
    vector<float> vu(Nx*dimVect);
    fillrandom(vu);
    float *u = vu.data();
    
    vector<float> vv(Ny*dimVect);
    fillrandom(vv);
    float *v = vv.data();

    // wrap variables
    vector<float*> vargs(4);
    vargs[0]=x;
    vargs[1]=y;
    vargs[2]=v;
    vargs[3]=u;
    float **args = vargs.data();

    float params[1];
    float Sigma = 1;
    params[0] = 1.0/(Sigma*Sigma);

    GpuConv1D(params, Nx, Ny, f, args);

}

extern "C" int GaussGpuGrad1Conv(float ooSigma2, float* alpha_h, float* x_h, float* y_h, float* beta_h, float* gamma_h, int dimPoint, int dimVect, int nx, int ny) ;

void main_specific(int Nx) {

    int Ny= Nx /2 ;

    int dimPoint = 3;
    int dimVect = 3;

    vector<float> vf(Nx*dimVect);
    fillrandom(vf);
    float *f = vf.data();

    vector<float> vx(Nx*dimPoint);
    fillrandom(vx);
    float *x = vx.data();
    
    vector<float> vy(Ny*dimPoint);
    fillrandom(vy);
    float *y = vy.data();
    
    vector<float> vu(Nx*dimVect);
    fillrandom(vu);
    float *u = vu.data();
    
    vector<float> vv(Ny*dimVect);
    fillrandom(vv);
    float *v = vv.data();
    
    float Sigma =1;
    float ooSigma2 = 1.0/(Sigma*Sigma);

    GaussGpuGrad1Conv(ooSigma2, u, x, y, v, f, 3,3,Nx,Ny);

}

/////////////////////////////////////////////////////////////////////////////////////
//                          Call the benchmark                                     //
/////////////////////////////////////////////////////////////////////////////////////


// The zeroth benchmark : simply to avoid warm up the GPU...
static void BM_dummy(benchmark::State& state) {
    for (auto _ : state)
        main_generic_2D(1000);
}
BENCHMARK(BM_dummy);// Register the function as a benchmark


// A first Benchmark:
static void cuda_specific(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_specific(Nx); 
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start); 
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_specific)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark

// A second one: 
static void cuda_generic_2D(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_generic_2D(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start); 
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_generic_2D)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark

// A third one:
static void cuda_generic_1D(benchmark::State& state) {
    int Nx = state.range(0);

    for (auto _ : state) {
        auto start = chrono::high_resolution_clock::now();
        //----------- the function to be benchmarked ------------//
        main_generic_1D(Nx);
        //------------------------------------------------------//
        auto end   = chrono::high_resolution_clock::now();

        auto elapsed_seconds = chrono::duration_cast<chrono::duration<double>>( end - start); 
        state.SetIterationTime(elapsed_seconds.count());
    }
}
// set range of the parameter to be tested : [ 8, 64, 512, 4k, 8k ]
BENCHMARK(cuda_generic_1D)->Range(8, 8<<10)->UseManualTime();// Register the function as a benchmark

BENCHMARK_MAIN();// generate the benchmarks
