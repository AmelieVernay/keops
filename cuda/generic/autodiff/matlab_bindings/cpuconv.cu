#include "hip/hip_runtime.h"

// see compile_mex file for compiling

#include <mex.h>
#include "core/CpuConv.cpp"
#include "core/autodiff.h"
#include "core/newsyntax.h"

// FORMULA and __TYPE__ are supposed to be set via #define macros in the compilation command

using F = decltype(FORMULA);

void ExitFcn(void) {
    hipDeviceReset();
}

class mystream : public std::streambuf
{
protected:
virtual std::streamsize xsputn(const char *s, std::streamsize n) { mexPrintf("%.*s", n, s); return n; }
virtual int overflow(int c=EOF) { if (c != EOF) { mexPrintf("%.1s", &c); } return 1; }
};
class scoped_redirect_cout
{
public:
	scoped_redirect_cout() { old_buf = std::cout.rdbuf(); std::cout.rdbuf(&mout); }
	~scoped_redirect_cout() { std::cout.rdbuf(old_buf); }
private:
	mystream mout;
	std::streambuf *old_buf;
};
static scoped_redirect_cout mycout_redirect;


//////////////////////////////////////////////////////////////////
///////////////// MEX ENTRY POINT ////////////////////////////////
//////////////////////////////////////////////////////////////////


/* the gateway function */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
    // register an exit function to prevent crash at matlab exit or recompiling
    mexAtExit(ExitFcn);
    
    const int TAG = 0;

	using VARSI = typename F::template VARS<TAG>;	// list variables of type I used in formula F
	using VARSJ = typename F::template VARS<1-TAG>; // list variables of type J used in formula F
	
	using DIMSX = GetDims<VARSI>;
	using DIMSY = GetDims<VARSJ>;
	
    using PARAMS = typename F::VARS<2>;
    static const int DIMPARAM = PARAMS::SIZE;

	using INDSI = GetInds<VARSI>;
	using INDSJ = GetInds<VARSJ>;

	using INDS = ConcatPacks<INDSI,INDSJ>;
	
	const int NARGSI = VARSI::SIZE; // number of I variables used in formula F
	const int NARGSJ = VARSJ::SIZE; // number of J variables used in formula F

    int argu = 0;
    //----- the first input arguments: info--------------//
	double *info;
	info = mxGetPr(prhs[argu]);
	if(mxGetM(prhs[argu])!=1 || mxGetN(prhs[argu])!=3)
		mexErrMsgTxt("first arg should be info (1x3)");
	argu++;
    int n[2]; // n[0] will be nx, n[1] will be ny;
	for(int k=0; k<2; k++)
		n[k] = mxGetN(prhs[argu+(int)info[k]]);
	int NARGS = info[2];
	
cout << "	NARGSI = " << NARGSI << endl;
cout << "	NARGSJ = " << NARGSJ << endl;

F::PrintId();
    
    /*  check for proper number of arguments */
    if(nrhs != 1+(DIMPARAM?1:0)+NARGS) // info, args..., params or info, args... if no parameter in formula
    {
        cout << "number of inputs is " << nrhs << endl;
        cout << "number of inputs should be " << 1+(DIMPARAM?1:0)+NARGS << endl;
        mexErrMsgTxt("Wrong number of inputs.");
    }
    if(nlhs != 1) 
        mexErrMsgTxt("One output required.");


    //////////////////////////////////////////////////////////////
    // Input arguments
    //////////////////////////////////////////////////////////////


    int *typeargs = new int[NARGS];
    int *dimargs = new int[NARGS];
    for(int k=0; k<NARGS; k++)
    {
        typeargs[k] = -1;
        dimargs[k] = -1;
    }
    for(int k=0; k<NARGSI; k++)
    {
        typeargs[INDSI::VAL(k)] = TAG;
        dimargs[INDSI::VAL(k)] = DIMSX::VAL(k);
    }
    for(int k=0; k<NARGSJ; k++)
    {
        typeargs[INDSJ::VAL(k)] = 1-TAG;
        dimargs[INDSJ::VAL(k)] = DIMSY::VAL(k);
    }

    //----- the next input arguments: args--------------//
    /*  create pointers to the input vectors */
    double **args = new double*[NARGS];    
    for(int k=0; k<NARGS; k++)
    {
    	/*  input sources */
    	args[k] = mxGetPr(prhs[argu+k]);
    	// checking dimensions
    	if(dimargs[k]!=-1) // we care only if the current variable is used in formula
    	{
			int dimk = mxGetM(prhs[argu+k]);
			// we check nx and ny here from the formula
			int nk = mxGetN(prhs[argu+k]);
			int typek = typeargs[k];
			cout << "k=" << k << endl;
			cout << "typek=" << typek << endl;
			cout << "nk=" << nk << endl;
			cout << "n[typek]=" << n[typek] << endl;
			cout << "dimk=" << dimk << endl;
			cout << "dimargs[k]=" << dimargs[k] << endl;
			// we check dimension here from the formula
			if(dimk!=dimargs[k])
				mexErrMsgTxt("wrong dimension for input");
			if(n[typek]!=nk)
			{
				mexErrMsgTxt("inconsistent input sizes");
			}
		}
    }
    
	double *params;
	if(DIMPARAM) {
		//----- the last input argument: params--------------//
		argu+=NARGS;
		/*  create a pointer to the input vector */
		params = mxGetPr(prhs[argu]);
		/*  get the dimensions of the input targets */
		int mp = mxGetM(prhs[argu]); //nrows
		int np = mxGetN(prhs[argu]); //ncols
		/* check to make sure the array is 1D */
cout << "min(mp,np) = " << min(mp,np) << endl;
		if( min(mp,np)!=1 )
			mexErrMsgTxt("Input params must be a 1D array.");
		np = max(mp,np);
		if(np!=DIMPARAM)
			mexErrMsgTxt("wrong dimension for input");
	}
	
    //////////////////////////////////////////////////////////////
    // Output arguments
    //////////////////////////////////////////////////////////////

    /*  set the output pointer to the output result(vector) */
    int dimout = F::DIM;
    int nout = n[TAG];
    plhs[0] = mxCreateDoubleMatrix(dimout,nout,mxREAL);

    /*  create a C pointer to a copy of the output result(vector)*/
    double *gamma = mxGetPr(plhs[0]);

    //////////////////////////////////////////////////////////////
    // Call Cuda codes
    //////////////////////////////////////////////////////////////
    
    CpuConv(Generic<F,TAG>::sEval(), params, n[TAG], n[1-TAG], gamma, args);

    delete[] args;
    delete[] typeargs;
    delete[] dimargs;
    

}
