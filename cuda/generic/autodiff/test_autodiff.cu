#include "hip/hip_runtime.h"
// test convolution with autodiff
// compile with 
//		nvcc -std=c++11 -O2 -o test_autodiff test_autodiff.cu

// we define an arbitrary function using available blocks,
// then test its convolution on the GPU, then get its gradient and test again the convolution

// Here we build the function F(x,y,u,v,beta) = <u,v>^2 * exp(-C*|x-y|^2) * beta
// where x, y, beta are 3D vectors, and u, v are 4D vectors
// and the convolution is gamma_i = sum_j F(x_i,y_j,u_i,v_j,beta_j)
// then we define G(x,y,u,v,beta,eta) = gradient of F with respect to x, with new input variable eta (3D)
// and the new convolution is gamma_i = sum_j G(x_i,y_j,u_i,v_j,beta_j,eta_i)

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ctime>
#include <algorithm>

#include "GpuConv2D.cu"
#include "autodiff.h"

using namespace std;



float floatrand() { return ((float)rand())/RAND_MAX-.5; } // random value between -.5 and .5

template < class V > void fillrandom(V& v) { generate(v.begin(), v.end(), floatrand); } // fills vector with random values

int main()
{
	// In this part we define the symbolic variables of the function
	using X = Var<0,3>; 	// X is the first variable and represents a 3D vector
	using Y = Var<1,3>; 	// Y is the second variable and represents a 3D vector
	using U = Var<2,4>; 	// U is the third variable and represents a 4D vector
	using V = Var<3,4>; 	// V is the fourth variable and represents a 4D vector
	using Beta = Var<4,3>;	// Beta is the fifth variable and represents a 3D vector
	using C = Param<0>;		// C is the first extra parameter
	
	// symbolic expression of the function
	// Available operations are :
	// 		IntConstant<N>				: constant integer function with value N
	// 		Constant<PRM>				: constant function with value given by parameter PRM (ex : Constant<C> here)
	// 		Add<FA,FB>					: adds FA and FB functions
	//		Scalprod<FA,FB> 			: scalar product between FA and FB
	//		Scal<FA,FB>					: product of FA (scalar valued) with FB
	//		SqNorm2<F>					: alias for Scalprod<F,F>
	//		Exp<F>						: exponential of F (scalar valued)
	//		Pow<F,M>					: Mth power of F (scalar valued) ; M is an integer
	//		Square<F>					: alias for Pow<F,2>
	//		Minus<F>					: alias for Scal<IntConstant<-1>,F>
	//		Subtract<FA,FB>				: alias for Add<FA,Minus<FB>>
	//		GaussKernel<PRM,FA,FB,FC> 	: alias for Scal<Exp<Scal<Constant<PRM>,Minus<SqNorm2<Subtract<FA,FB>>>>>,FC>
	//		Grad<F,V,GRADIN>			: gradient (in fact transpose of diff op) of F with respect to variable V, applied to GRADIN
	
	// here we define F = <U,V>^2 * exp(-C*|X-Y|^2) * Beta in usual notations
	using F = Scal<Square<Scalprod<U,V>>,GaussKernel<C,X,Y,Beta>>;

	// precise which variables will be indexed by i or j when calling the convolution with function F
	using FVARSI = univpack<X,U>;
	using FVARSJ = univpack<Y,V,Beta>;
	const int FDIMPARAM = 1; // there is only one parameter 
	
	using FUNCONVF = typename Generic<F,FVARSI,FVARSJ,FDIMPARAM>::sEval;
	
	// gradient with respect to X	
	using Eta = Var<5,F::DIM>;	// new variable is in sixth position and is input of gradient
	using G = Grad<F,X,Eta>;
	
	// precise i and j variables for convolution with G
	using GVARSI = univpack<Eta,X,U>;
	using GVARSJ = univpack<Y,V,Beta>;
	const int GDIMPARAM = 1;

	using FUNCONVG = typename Generic<G,GVARSI,GVARSJ,GDIMPARAM>::sEval;
	
	
	
	
	
	// now we test
	
	int Nx=5000, Ny=5000;

	vector<float> vf(Nx*F::DIM); fillrandom(vf); float *f = vf.data();
	vector<float> vx(Nx*X::DIM); fillrandom(vx); float *x = vx.data();
	vector<float> vy(Ny*Y::DIM); fillrandom(vy); float *y = vy.data();
	vector<float> vu(Nx*U::DIM); fillrandom(vu); float *u = vu.data();
	vector<float> vv(Ny*V::DIM); fillrandom(vv); float *v = vv.data();
	vector<float> vb(Ny*Beta::DIM); fillrandom(vb); float *b = vb.data();
	
	vector<float> resgpu(Nx*F::DIM), rescpu(Nx*F::DIM);
	
	float params[1];
	float Sigma = 1;
	params[0] = 1.0/(Sigma*Sigma);
	
	clock_t begin, end;
	
	begin = clock();
	int deviceID = 0;
	hipSetDevice(deviceID);
	end = clock();
	cout << "time for GPU initialization : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	cout << "testing function F" << endl;
	begin = clock();
	GpuConv2D(FUNCONVF(), params, Nx, Ny, f, x, y, u, v, b); 
	end = clock();
	cout << "time for GPU computation (first run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	begin = clock();
	GpuConv2D(FUNCONVF(), params, Nx, Ny, f, x, y, u, v, b); 
	end = clock();
	cout << "time for GPU computation (second run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	resgpu = vf;
		
	begin = clock();
	CpuConv(FUNCONVF(), params, Nx, Ny, f, x, y, u, v, b); 
	end = clock();
	cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	rescpu = vf;
	
	// display mean of errors
	float s = 0;
	for(int i=0; i<Nx*F::DIM; i++)
		s += abs(resgpu[i]-rescpu[i]);
	cout << "mean abs error =" << s/Nx << endl;





	vector<float> ve(Nx*Eta::DIM); fillrandom(ve); float *e = ve.data();

	cout << "testing function G" << endl;
	begin = clock();
	GpuConv2D(FUNCONVG(), params, Nx, Ny, f, x, y, u, v, b, e); 
	end = clock();
	cout << "time for GPU computation (first run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	begin = clock();
	GpuConv2D(FUNCONVG(), params, Nx, Ny, f, x, y, u, v, b, e); 
	end = clock();
	cout << "time for GPU computation (second run) : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	resgpu = vf;
		
	begin = clock();
	CpuConv(FUNCONVG(), params, Nx, Ny, f, x, y, u, v, b, e); 
	end = clock();
	cout << "time for CPU computation : " << double(end - begin) / CLOCKS_PER_SEC << endl;
	
	rescpu = vf;
	
	// display mean of errors
	s = 0;
	for(int i=0; i<Nx*G::DIM; i++)
		s += abs(resgpu[i]-rescpu[i]);
	cout << "mean abs error =" << s/Nx << endl;


}



