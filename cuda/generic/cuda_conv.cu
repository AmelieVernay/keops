#include "hip/hip_runtime.h"

// nvcc -std=c++11 -Xcompiler -fPIC -shared -o cuda_conv.so cuda_conv.cu

#include "GpuConv2D.cu"

#define CALL_SCALARRADIALKER(TYPE,DIMPOINT,DIMVECT) \
	if(dimPoint==DIMPOINT && dimVect==DIMVECT) \
	{ \
		typedef ScalarRadialKernel<TYPE,DIMPOINT,DIMVECT,RadialFun> Ker; \
		struct Ker::sEval funeval; \
		return GpuConv2D(Ker(RadialFun(Sigma)),funeval, nx, ny, gamma_h, x_h, y_h, beta_h); \
	}

// here we give the list of possible values for DIMVECT
#define CALL_SCALARRADIALKER_DIMVECT(TYPE,DIMPOINT) \
	CALL_SCALARRADIALKER(TYPE,DIMPOINT,1) \
	CALL_SCALARRADIALKER(TYPE,DIMPOINT,2) \
	CALL_SCALARRADIALKER(TYPE,DIMPOINT,3)
	
// here we give the list of possible values for DIMPOINT
#define CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	CALL_SCALARRADIALKER_DIMVECT(TYPE,1) \
	CALL_SCALARRADIALKER_DIMVECT(TYPE,2) \
	CALL_SCALARRADIALKER_DIMVECT(TYPE,3)


#define DECLARE_EXTERNC_SCALARRADIAL_Gauss(TYPE) \
extern "C" int GaussGpuEvalConv(TYPE ooSigma2, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int dimPoint, int dimVect, int nx, int ny) \
{ \
	TYPE Sigma = sqrt(1/ooSigma2); \
	typedef GaussFunction<TYPE> RadialFun; \
	CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	cout << "These dimensions are not implemented, but you just need to copy-paste one line and recompile." << endl; \
	return -1; \
}

#define DECLARE_EXTERNC_SCALARRADIAL_Cauchy(TYPE) \
extern "C" int CauchyGpuEvalConv(TYPE ooSigma2, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int dimPoint, int dimVect, int nx, int ny) \
{ \
	TYPE Sigma = sqrt(1/ooSigma2); \
	typedef CauchyFunction<TYPE> RadialFun; \
	CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	cout << "These dimensions are not implemented, but you just need to copy-paste one line and recompile." << endl; \
	return -1; \
}

#define DECLARE_EXTERNC_SCALARRADIAL_Laplace(TYPE) \
extern "C" int LaplaceGpuEvalConv(TYPE ooSigma2, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int dimPoint, int dimVect, int nx, int ny) \
{ \
	TYPE Sigma = sqrt(1/ooSigma2); \
	typedef LaplaceFunction<TYPE> RadialFun; \
	CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	cout << "These dimensions are not implemented, but you just need to copy-paste one line and recompile." << endl; \
	return -1; \
}

#define DECLARE_EXTERNC_SCALARRADIAL_Energy(TYPE) \
extern "C" int EnergyGpuEvalConv(TYPE ooSigma2, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int dimPoint, int dimVect, int nx, int ny) \
{ \
	TYPE Sigma = sqrt(1/ooSigma2); \
	typedef EnergyFunction<TYPE> RadialFun; \
	CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	cout << "These dimensions are not implemented, but you just need to copy-paste one line and recompile." << endl; \
	return -1; \
}

#define DECLARE_EXTERNC_SCALARRADIAL_Sum4Gauss(TYPE) \
extern "C" int Sum4GaussGpuEvalConv(TYPE ooSigma2, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int dimPoint, int dimVect, int nx, int ny) \
{ \
	TYPE Sigma = sqrt(1/ooSigma2); \
	typedef Sum4GaussFunction<TYPE> RadialFun; \
	CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	cout << "These dimensions are not implemented, but you just need to copy-paste one line and recompile." << endl; \
	return -1; \
}

#define DECLARE_EXTERNC_SCALARRADIAL_Sum4Cauchy(TYPE) \
extern "C" int Sum4CauchyGpuEvalConv(TYPE ooSigma2, TYPE* x_h, TYPE* y_h, TYPE* beta_h, TYPE* gamma_h, int dimPoint, int dimVect, int nx, int ny) \
{ \
	TYPE Sigma = sqrt(1/ooSigma2); \
	typedef Sum4CauchyFunction<TYPE> RadialFun; \
	CALL_SCALARRADIALKER_DIMPOINT_DIMVECT(TYPE) \
	cout << "These dimensions are not implemented, but you just need to copy-paste one line and recompile." << endl; \
	return -1; \
}

#if !(UseCudaOnDoubles) 
	DECLARE_EXTERNC_SCALARRADIAL_Gauss(double)
	DECLARE_EXTERNC_SCALARRADIAL_Cauchy(double)
	DECLARE_EXTERNC_SCALARRADIAL_Sum4Gauss(double)
	DECLARE_EXTERNC_SCALARRADIAL_Sum4Cauchy(double)
#else
	DECLARE_EXTERNC_SCALARRADIAL_Gauss(float)
	DECLARE_EXTERNC_SCALARRADIAL_Cauchy(float)
	DECLARE_EXTERNC_SCALARRADIAL_Sum4Gauss(float)
	DECLARE_EXTERNC_SCALARRADIAL_Sum4Cauchy(float)
#endif
